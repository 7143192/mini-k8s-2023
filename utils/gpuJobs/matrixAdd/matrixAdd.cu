// #include <stdio.h>
// #include "cuda_runtime.h"
// #include "device_launch_parameters.h"
// #include <cuda_runtime.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

#define CHECK(call)                                                    \
  {                                                                    \
    const hipError_t error = call;                                    \
    if (error != hipSuccess) {                                        \
      printf("ERROR: %s:%d,", __FILE__, __LINE__);                     \
      printf("code:%d,reason:%s\n", error, hipGetErrorString(error)); \
      exit(1);                                                         \
    }                                                                  \
  }

// init device.(device <--> running on GPU. )
void initDevice(int devNum) {
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
}

// init the matrix.
void initMatrix(int* matrix, int size) {
  for (int i = 0; i < size; ++i) {
    matrix[i] = i + 1;
  }
}

// print out final result to DEBUG
//NOTE: in lyh's VM, there is NO GPU Device to debug....
void printMatrix(int* mat, int nBytes){
  printf("matrixAdd running res = \n");
  for(int i=0; i < nBytes; i++){
    printf("%d ", mat[i]);
    if (i != 0) {
      if (i % 1023 == 0) {
        printf("\n");
      }
    }
  }
}

__global__
void matrixAdd(int* A, int* B, int* res, int nx,int ny) {
  int ix = threadIdx.x + blockDim.x * blockIdx.x;
  int iy = threadIdx.y + blockDim.y * blockIdx.y;
  int idx = ix + iy * ny;
  if (ix >= nx || iy >= ny) {
    return ;
  }
  res[idx] = A[idx] + B[idx];
}

int main(int argc, char** argv) {
  //init dev
  initDevice(0);

  int nx = 1 << 10;
  int ny = 1 << 10;
  int nBytes = nx * ny * sizeof(int);
  // variables used in HOST device.
  int* A_host = (int*) malloc(nBytes);
  int* B_host = (int*) malloc(nBytes);
  int* host_res = (int*) malloc(nBytes);
  initMatrix(A_host, nx * ny);
  initMatrix(B_host, nx * ny);
  // variables used in DEVICE(GPU) side.
  int* A_dev = NULL;
  int* B_dev = NULL;
  int* dev_res = NULL;
  // use function cudaMalloc(void**, int) to malloc in GPU side.
  CHECK(hipMalloc((void**)&A_dev, nBytes));
  CHECK(hipMalloc((void**)&B_dev, nBytes));
  CHECK(hipMalloc((void**)&dev_res, nBytes));
  // use function cudaMemcpy(void* dst, const void* src, size_t count, cudaMemcpyKind kind) to copy var from GPU back to HOST(CPU).
  CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));
  // block: 16 * 16
  dim3 threadsPerBlock(16, 16);
  // NOTE: do not forget "+1" operation!
  // grid size : 64 * 64
  dim3 numBlocks((nx - 1) / threadsPerBlock.x + 1, (ny - 1) / threadsPerBlock.y + 1);
  // call the __global__ function to add every position of the two matrixes parallelly.
  matrixAdd<<<numBlocks, threadsPerBlock>>>(A_dev, B_dev, dev_res, nx, ny);
  // use the function sync to wait all functions finish.
  CHECK(hipDeviceSynchronize());
  // copy final result of MatrixAdd from GPU to CPU.
  CHECK(hipMemcpy(host_res, dev_res, nBytes, hipMemcpyDeviceToHost));
  // used to DEBUG.
  printMatrix(host_res, nx * ny);

  // use function CudaFree() to free var in GPU side.
  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(dev_res);
  // normal free function in CPU side.
  free(A_host);
  free(B_host);
  free(host_res);
  // cudaDeviceReset();
  return 0;
}